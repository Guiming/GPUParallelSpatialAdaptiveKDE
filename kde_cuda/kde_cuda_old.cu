#include "hip/hip_runtime.h"
// Copyright 2016 Guiming Zhang (gzhang45@wisc.edu)
// Distributed under GNU General Public License (GPL) license

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

#include "SamplePoints.h"
#include "AsciiRaster.h"
#include "Utilities.h"

#include "kde_kernel_old.cu"

using namespace std;

// distance squared between two points
inline  float Distance2(float x0, float y0, float x1, float y1){
	float dx = x1 - x0;
	float dy = y1 - y0;
	return dx*dx + dy*dy;
}

// mean center of points
void MeanCenter(SamplePoints Points, float &mean_x, float &mean_y);

// (squared) standard distance of points
void StandardDistance2(SamplePoints Points, float &d2);

// bandwidth squared
inline float BandWidth2(SamplePoints Points){
	float d2;
	StandardDistance2(Points, d2);
	return sqrtf(2.0f / (3 * Points.numberOfPoints)) * d2;
}

// Gaussian kernel
inline float GaussianKernel(float h2, float d2){
	return expf(d2 / (-2.0f * h2)) / (h2*TWO_PI);
}

SamplePoints AllocateDeviceSamplePoints(const SamplePoints Points);
void CopyToDeviceSamplePoints(SamplePoints dPoints, const SamplePoints hPoints);
SamplePoints AllocateSamplePoints(int n); // random points
SamplePoints ReadSamplePoints(const char *csvFile); // points read from a .csv file
void FreeDeviceSamplePoints(SamplePoints* dPoints);
void FreeSamplePoints(SamplePoints* Points);
void WriteSamplePoints(SamplePoints* Points, const char * csvFile);
void WriteSamplePoints(SamplePoints* Points, float* Hs, float* Ws, const char * csvFile);

AsciiRaster AllocateDeviceAsciiRaster(const AsciiRaster Ascii);
void CopyToDeviceAsciiRaster(AsciiRaster dAscii, const AsciiRaster hAscii);
void CopyFromDeviceAsciiRaster(AsciiRaster hAscii, const AsciiRaster dAscii);
AsciiRaster AllocateAsciiRaster(int nCols, int nRows, float xLLCorner, float yLLCorner, float cellSize, float noDataValue);
AsciiRaster ReadAsciiRaster(char * asciiFile); // ascii raster read from a .asc file
AsciiRaster CopyAsciiRaster(const AsciiRaster Ascii);
void FreeDeviceAsciiRaster(AsciiRaster* Ascii);
void FreeAsciiRaster(AsciiRaster* Ascii);
void WriteAsciiRaster(AsciiRaster* Ascii, const char * asciiFile);

float* AllocateEdgeCorrectionWeights(SamplePoints Points);
void FreeEdgeCorrectionWeights(float* weights);

float* AllocateDeviceEdgeCorrectionWeights(SamplePoints Points);
void FreeDeviceEdgeCorrectionWeights(float* weights);

///////// Guiming on 2016-03-16 ///////////////
// the array holding bandwidth at each point
float* AllocateBandwidths(int n); // n is number of points
float* AllocateDeviceBandwidths(int n); // n is number of points
void CopyToDeviceBandwidths(float* dBandwidth, const float* hBandwidths, const int n);
void CopyFromDeviceBandwidths(float* hBandwidth, const float* dBandwidths, const int n);
void FreeDeviceBandwidths(float* bandwidths);
void FreeBandwidths(float* bandwidths);

// the array holding inclusive/exclusive density at each point
float* AllocateDen(int n); // n is number of points
float* AllocateDeviceDen(int n); // n is number of points
void CopyToDeviceDen(float* dDen, const float* hDen, const int n);
void CopyFromDeviceDen(float* hDen, const float* dDen, const int n);
void CopyDeviceDen(float* dDenTo, const float* dDenFrom, const int n);
void FreeDeviceDen(float* den);
void FreeDen(float* den);

// compute the optimal Maximum Likelihood Estimation fixed bandwidth
// By Guiming @ 2016-02-26
float MLE_FixedBandWidth(AsciiRaster Ascii, SamplePoints Points, float *edgeWeights, float h, float* den0 = NULL, float* den1 = NULL, bool useGPU = false, float* dDen0 = NULL, float* dDen1 = NULL);

// compute fixed bandwidth density at sample points
// By Guiming @ 2016-05-21
void ComputeFixedDenistyAtPoints(AsciiRaster Ascii, SamplePoints Points, float *edgeWeights, float h, float* den0 = NULL, float* den1 = NULL, bool useGPU = false, float* dDen0 = NULL, float* dDen1 = NULL);

// compute the log likelihood given single bandwidth h
// By Guiming @ 2016-02-26
float LogLikelihood(AsciiRaster Ascii, SamplePoints Points, float *edgeWeights, float h, float* den0 = NULL, float* den1 = NULL, bool useGPU = false, float* dDen0 = NULL, float* dDen1 = NULL);

// compute the log likelihood given bandwidths hs
// By Guiming @ 2016-02-26
// float* den0 : density based on all points, including itself
// float* den1 : leave one out density
float LogLikelihood(AsciiRaster Ascii, SamplePoints Points, float *edgeWeights, float* hs, float* den0 = NULL, float* den1 = NULL, bool useGPU = false, float* dHs = NULL, float* dDen0 = NULL, float* dDen1 = NULL, float h = 1.0f, float alpha = -0.5f, float* dDen0cpy = NULL);

// compute the log likelihood given a center (h0, alpha0) and step (stepH, stepA)
// By Guiming @ 2016-03-06
void hj_likelihood(AsciiRaster Ascii, SamplePoints Points, float *edgeWeights, float h0, float alpha0, float stepH, float stepA, int lastdmax, float* logLs, float* hs = NULL, float* den0 = NULL, float* den1 = NULL, bool useGPU = false, float* dHs = NULL, float* dDen0 = NULL, float* dDen1 = NULL, float* dDen0cpy = NULL);

// compute the optimal h and alpha (parameters for calculating the optimal adaptive bandwith)
// By Guiming @ 2016-03-06
void hooke_jeeves(AsciiRaster Ascii, SamplePoints Points, float *edgeWeights, float h0, float alpha0, float stepH, float stepA, float* optParas, float* hs = NULL, float* den0 = NULL, float* den1 = NULL, bool useGPU = false, float* dHs = NULL, float* dDen0 = NULL, float* dDen1 = NULL, float* dDen0cpy = NULL);

float compGML(float* den0, int n);
///////// Guiming on 2016-03-16 ///////////////


// exact edge effects correction (Diggle 1985)
void EdgeCorrectionWeightsExact(SamplePoints Points, float h, AsciiRaster Ascii, float *weights);
void EdgeCorrectionWeightsExact(SamplePoints Points, float *hs, AsciiRaster Ascii, float *weights);

// approximate edge effects correction (Diggle 1985) (not adopted)
AsciiRaster EdgeCorrectionWeightsApprox(AsciiRaster Ascii, float h2);

// check whether the result from sequential computation and that from parallel computation agree
void CheckResults(AsciiRaster AsciiSEQ, AsciiRaster AsciiPARA);

// reduction an array on GPU
void ReductionSumGPU(float* dArray, int numberOfElements);

/* Run in 2 modes
 *
 * Mode 0: Do not read points and mask from files.
 *         User specify # of points and cell size of the estimated intensity surface.
 *         Random points with x, y coordinates in the range [0,100] will be generated.
 *         The cell size (must be less than 100) determines how many cells in the intensity surface raster.
 *
 *         ./kde_cuda [mode] [#points] [cellsize] [skipSEQ] [skipPARA]
 *         e.g., ./kde_cuda 0 100 1.0 0 0
 *
 * Mode 1: Read points and mask from files.
 *
 *         ./kde_cuda [mode] [points_file] [mask_file] [skipSEQ] [skipPARA]
 *         e.g., ./kde_cuda 1 ../Points.csv ../Mask.asc 0 0
 *
*/

int main(int argc, char *argv[]){

	int NPNTS = 100;                // default # of points
	float CELLSIZE = 1.0f;          // default cellsize
	char* pntFn = "data/Points.csv";  // default points file
	char* maskFn = "data/Mask.asc";   // default mask file
	bool fromFiles = true;          // by default, read Points and Mask from files

	int SKIPSEQ = 0;                // by default, do not skip sequential execution
	int SKIPPARA = 0;               // by default, do not skip parallel execution

	//Guiming May 1, 2016
	int Hoption = 0; // 0 for rule of thumb
					 // 1 for h optimal
					 // 2 for h adaptive
	char* denSEQfn = "data/den_SEQ.asc";
	char* denCUDAfn = "data/den_CUDA.asc";

	// parse commandline arguments
	if(argc != 9){
		printf("Incorrect arguments provided. Exiting...\n");
		printf("Run in mode 0:\n ./kde_cuda 0 #points cellsize h_option skip_sequential skip_parallel denfn_seq, denfn_cuda\n");
		printf("Run in mode 1:\n ./kde_cuda 1 points_file mask_file h_option skip_sequential skip_parallel denfn_seq, denfn_cuda\n");
        return 1;
	}
	else{
		int mode = atoi(argv[1]);
		if(mode == 0){
			fromFiles = false;
			NPNTS = atoi(argv[2]);
			CELLSIZE = (float)atof(argv[3]);
			Hoption = atoi(argv[4]);
			SKIPSEQ = atoi(argv[5]);
			SKIPPARA = atoi(argv[6]);
			denSEQfn = argv[7];
			denCUDAfn = argv[8];
		}
		else if(mode == 1){
			pntFn = argv[2];
			maskFn = argv[3];
			Hoption = atoi(argv[4]);
			SKIPSEQ = atoi(argv[5]);
			SKIPPARA = atoi(argv[6]);
			denSEQfn = argv[7];
			denCUDAfn = argv[8];
		}
		else{
			printf("Incorrect arguments provided. Exiting...\n");
			printf("Run in mode 0:\n ./kde_cuda 0 #points cellsize h_option skip_sequential skip_parallel denfn_seq, denfn_cuda\n");
			printf("Run in mode 1:\n ./kde_cuda 1 points_file mask_file h_option skip_sequential skip_parallel denfn_seq, denfn_cuda\n");
	        return 1;
		}

	}

	SamplePoints Points; // sample of point events
	AsciiRaster Mask;    // a mask indicating the extent of study area
	AsciiRaster DenSurf, DenSurf_CUDA; // the estimated intensity surface
	float *edgeWeights;  // edge effect correct weights (for each point in the sample)

	bool correction = true; // enable edge effect correction

	srand(100); // If not read from files, generate random points

	if (fromFiles){
		Points = ReadSamplePoints(pntFn);
		Mask = ReadAsciiRaster(maskFn);
	}
	else{
		Points = AllocateSamplePoints(NPNTS);
		Mask = AllocateAsciiRaster(int(100/CELLSIZE), int(100/CELLSIZE), 0.0f, 0.0f, CELLSIZE, -9999.0f);
	}
	DenSurf = CopyAsciiRaster(Mask);

	// parameters
	int numPoints = Points.numberOfPoints;
	int nCols = Mask.nCols;
	int nRows = Mask.nRows;
	float xLLCorner = Mask.xLLCorner;
	float yLLCorner = Mask.yLLCorner;
	float noDataValue = Mask.noDataValue;
	float cellSize = Mask.cellSize;

	printf("number of points: %d\n", numPoints);
	printf("cell size: %f\n", cellSize);
	printf("number of cells: %d\n", nCols * nRows);

	printf("skip executing SEQUENTIAL program? %d\n", SKIPSEQ);
	printf("skip executing PARALLEL program? %d\n", SKIPPARA);
	printf("number of threads per block: %d\n", BLOCK_SIZE);

	// do the work
	float cell_x; // x coord of cell
	float cell_y; // y coord of cell
	float p_x;    // x coord of point
	float p_y;    // x coord of point
	float p_w;    // weight of point
	float e_w = 1.0;    // edge effect correction weight

	float h = sqrtf(BandWidth2(Points));
	printf("rule of thumb bandwidth h0: %.5f\n", h);

	// timing
	//double start, stop;
	float elaps_seq, elaps_exc, elaps_inc;
	hipError_t error;

	if(SKIPSEQ == 0){
		edgeWeights = NULL;
		edgeWeights = AllocateEdgeCorrectionWeights(Points);

	///////////////////////// SEQUENTIAL /////////////////////////////////

		///////////////////////// START CPU TIMING /////////////////////////////
		hipEvent_t startCPU;
		error = hipEventCreate(&startCPU);

		if (error != hipSuccess)
		{
		   printf("Failed to create start event (error code %s)!\n", hipGetErrorString(error));
		   exit(EXIT_FAILURE);
		}

		hipEvent_t stopCPU;
		error = hipEventCreate(&stopCPU);

		if (error != hipSuccess)
		{
		   printf("Failed to create stop event (error code %s)!\n", hipGetErrorString(error));
		   exit(EXIT_FAILURE);
		}

		// Record the start event
		error = hipEventRecord(startCPU, NULL);
		if (error != hipSuccess)
		{
		   printf("Failed to record start event (error code %s)!\n", hipGetErrorString(error));
		   exit(EXIT_FAILURE);
		}
		///////////////////////// END OF START CPU TIMING /////////////////////////////

		float* hs = AllocateBandwidths(numPoints);
		for(int i = 0; i < numPoints; i++){
			hs[i] = h;
		}

	    // compute edge effect correction weights
		EdgeCorrectionWeightsExact(Points, h, Mask, edgeWeights);

		if(Hoption == 1){
			float hopt = MLE_FixedBandWidth(Mask, Points, edgeWeights, h, NULL, NULL, false);
			printf("cross validated optimal fixed bandwidth hopt: %.5f\n", hopt);

			for(int i = 0; i < numPoints; i++){
				hs[i] = hopt;
			}

			// update edge correction weights
			if(UPDATEWEIGHTS){
				EdgeCorrectionWeightsExact(Points, hs, Mask, edgeWeights);
			}
		}

		if(Hoption == 2){
			float* den0 = AllocateDen(numPoints);
			float* den1 = AllocateDen(numPoints);
			float h0 = h;
			float alpha0 = -0.5;
			float stepH = h0/10;
			float stepA = 0.1;
			float* optParas = (float*)malloc(3*sizeof(float));
			hooke_jeeves(Mask, Points, edgeWeights, h0, alpha0, stepH, stepA, optParas, hs, den0, den1, false);
			h0 = optParas[0];
			alpha0 = optParas[1];
			float logL = optParas[2];

			if(DEBUG) printf("h0: %.5f alpha0: %.5f Lmax: %.5f\n", h0, alpha0, logL);

			free(optParas);
			optParas = NULL;

			ComputeFixedDenistyAtPoints(Mask, Points, edgeWeights, h0, den0, NULL, false);
			float gml = compGML(den0, numPoints);
			for(int i = 0; i < numPoints; i++){
				hs[i] = h0 * powf(den0[i]/gml, alpha0);
			}
			FreeDen(den0);
			FreeDen(den1);

			// update edge correction weights
			if(UPDATEWEIGHTS){
				EdgeCorrectionWeightsExact(Points, hs, Mask, edgeWeights);
			}
		}

		// KDE
		for (int row = 0; row < nRows; row++){
			cell_y = ROW_TO_YCOORD(row, nRows, yLLCorner, cellSize);
			for (int col = 0; col < nCols; col++){
				cell_x = COL_TO_XCOORD(col, xLLCorner, cellSize);
				int idx = row * nCols + col;
				if (DenSurf.elements[idx] != noDataValue){

					float den = 0.0;
					float hp;
					for (int p = 0; p < numPoints; p++){
						p_x = Points.xCoordinates[p];
						p_y = Points.yCoordinates[p];
						p_w = Points.weights[p];
						hp = hs[p];
						if (correction){
							e_w = edgeWeights[p];
						}
						float d2 = Distance2(p_x, p_y, cell_x, cell_y);
						den += GaussianKernel(hp * hp, d2) * p_w *e_w;
					}
					DenSurf.elements[idx] = den; // intensity, not probability
				}
			}
		}



		///////////////////////// STOP CPU TIMING /////////////////////////////
	    // Record the stop event
	    error = hipEventRecord(stopCPU, NULL);

	    if (error != hipSuccess)
	    {
	        printf("Failed to record stop event (error code %s)!\n", hipGetErrorString(error));
	        exit(EXIT_FAILURE);
	    }

	    // Wait for the stop event to complete
	    error = hipEventSynchronize(stopCPU);
	    if (error != hipSuccess)
	    {
	        printf("Failed to synchronize on the stop event (error code %s)!\n", hipGetErrorString(error));
	        exit(EXIT_FAILURE);
	    }

	    elaps_seq = 0.0f;
	    error = hipEventElapsedTime(&elaps_seq, startCPU, stopCPU);

	    if (error != hipSuccess)
	    {
	        printf("Failed to get time elapsed between events (error code %s)!\n", hipGetErrorString(error));
	        exit(EXIT_FAILURE);
	    }
	    ///////////////////////// END OF STOP CPU TIMING /////////////////////////////
		printf("Computation on CPU took %.3f ms\n", elaps_seq);

		// write results to file
		WriteAsciiRaster(&DenSurf, denSEQfn);
		WriteSamplePoints(&Points, hs, edgeWeights, "pntsSEQ.csv");

		// clean up (only those not needed any more)
		FreeEdgeCorrectionWeights(edgeWeights);
		//FreeAsciiRaster(&DenSurf);
		FreeBandwidths(hs);
	}
////////////////////////// END OF SEQUENTIAL //////////////////////////////


//////////////////////////  CUDA  /////////////////////////////////////////
	if(SKIPPARA == 0){
		DenSurf_CUDA = CopyAsciiRaster(Mask);
		SamplePoints dPoints = AllocateDeviceSamplePoints(Points);
		float* dWeights = AllocateDeviceEdgeCorrectionWeights(Points);
		AsciiRaster dAscii = AllocateDeviceAsciiRaster(Mask);

		// Guiming @ 2016-03-17
		float* hs = AllocateBandwidths(Points.numberOfPoints);
		for(int i = 0; i < numPoints; i++){
			hs[i] = h;
		}
		float* dHs = AllocateDeviceBandwidths(Points.numberOfPoints);

		float* den0 = AllocateDen(Points.numberOfPoints);
		float* dDen0 = AllocateDeviceDen(Points.numberOfPoints);
		float* dDen0cpy = AllocateDeviceDen(Points.numberOfPoints);

		float* den1 = AllocateDen(Points.numberOfPoints);
		float* dDen1 = AllocateDeviceDen(Points.numberOfPoints);

		///////////////////////// START GPU INCLUSIVE TIMING /////////////////////////////
		hipEvent_t startInc;
		error = hipEventCreate(&startInc);

		if (error != hipSuccess)
		{
		   printf("Failed to create start event (error code %s)!\n", hipGetErrorString(error));
		   exit(EXIT_FAILURE);
		}

		hipEvent_t stopInc;
		error = hipEventCreate(&stopInc);

		if (error != hipSuccess)
		{
		   printf("Failed to create stop event (error code %s)!\n", hipGetErrorString(error));
		   exit(EXIT_FAILURE);
		}

		// Record the start event
		error = hipEventRecord(startInc, NULL);
		if (error != hipSuccess)
		{
		   printf("Failed to record start event (error code %s)!\n", hipGetErrorString(error));
		   exit(EXIT_FAILURE);
		}
		///////////////////////// END OF START GPU INCLUSIVE TIMING /////////////////////////////
		CopyToDeviceBandwidths(dHs, hs, Points.numberOfPoints);
		CopyToDeviceSamplePoints(dPoints, Points);
		CopyToDeviceAsciiRaster(dAscii, Mask);

		///////////////////////// START GPU EXCLUSIVE TIMING /////////////////////////////
		hipEvent_t startExc;
		error = hipEventCreate(&startExc);

		if (error != hipSuccess)
		{
		   printf("Failed to create start event (error code %s)!\n", hipGetErrorString(error));
		   exit(EXIT_FAILURE);
		}

		hipEvent_t stopExc;
		error = hipEventCreate(&stopExc);

		if (error != hipSuccess)
		{
		   printf("Failed to create stop event (error code %s)!\n", hipGetErrorString(error));
		   exit(EXIT_FAILURE);
		}

		// Record the start event
		error = hipEventRecord(startExc, NULL);
		if (error != hipSuccess)
		{
		   printf("Failed to record start event (error code %s)!\n", hipGetErrorString(error));
		   exit(EXIT_FAILURE);
		}
		///////////////////////// END OF START GPU EXLUSIVE TIMING /////////////////////////////

		// invoke kernels to compute edge effect correction weights (for each point)
		// execution config.
		int NBLOCK_W = (dPoints.numberOfPoints + BLOCK_SIZE - 1) / BLOCK_SIZE;
	    int GRID_SIZE_W = (int)(sqrtf(NBLOCK_W)) + 1;
	    dim3 dimGrid_W(GRID_SIZE_W, GRID_SIZE_W);

		CalcEdgeCorrectionWeights<<<dimGrid_W, BLOCK_SIZE>>>(h * h, dPoints, dAscii, dWeights);

		// Guiming @ 2016-03-17
		/////////////////////////////////////////////////////////////////////////////////////////
		int numPoints = Points.numberOfPoints;

		if(Hoption == 1){
			float hopt = MLE_FixedBandWidth(dAscii, dPoints, dWeights, h, NULL, den1, true, NULL, dDen1);
			printf("cross validated optimal fixed bandwidth hopt: %.5f\n", hopt);

			// kind of combusome
			CalcVaryingBandwidths<<<dimGrid_W, BLOCK_SIZE>>>(dPoints, hopt, dHs);
			if(UPDATEWEIGHTS){
				CalcEdgeCorrectionWeights<<<dimGrid_W, BLOCK_SIZE>>>(dHs, dPoints, dAscii, dWeights);
			}
		}

		if(Hoption == 2){
			float h0 = h;
			float alpha0 = -0.5;
			float stepH = h0/10;
			float stepA = 0.1;
			float* optParas = (float*)malloc(3*sizeof(float));
			hooke_jeeves(dAscii, dPoints, dWeights, h0, alpha0, stepH, stepA, optParas, hs, den0, den1, true, dHs, dDen0, dDen1, dDen0cpy);
			h0 = optParas[0];
			alpha0 = optParas[1];
			float logL = optParas[2];

			if(DEBUG) printf("h0: %.5f alpha0: %.5f Lmax: %.5f\n", h0, alpha0, logL);

			free(optParas);
			optParas = NULL;

			ComputeFixedDenistyAtPoints(dAscii, dPoints, dWeights, h0, NULL, NULL, true, dDen0, NULL);
			CopyDeviceDen(dDen0cpy, dDen0, numPoints);
			ReductionSumGPU(dDen0cpy, numPoints);
			//float tmp = 0.0f;
			//hipMemcpyFromSymbol(&tmp, HIP_SYMBOL(dReductionSum), sizeof(float), 0, hipMemcpyDeviceToHost);
			//printf("reduction result (geometricmean): %3.4f \n", tmp);

	    	// update bandwidth on GPU
	    	CalcVaryingBandwidths<<<dimGrid_W, BLOCK_SIZE>>>(Points, dDen0, h0, alpha0, dHs);

			// update weights
			//CopyToDeviceBandwidths(dHs, hs, numPoints);
			if(UPDATEWEIGHTS){
				CalcEdgeCorrectionWeights<<<dimGrid_W, BLOCK_SIZE>>>(dHs, dPoints, dAscii, dWeights);
			}
		}

		/////////////////////////////////////////////////////////////////////////////////

		// invoke kernel to do density estimation
		int NBLOCK_K = (dAscii.nCols*dAscii.nRows + BLOCK_SIZE - 1) / BLOCK_SIZE;
	    int GRID_SIZE_K = (int)(sqrtf(NBLOCK_K)) + 1;
	    dim3 dimGrid_K(GRID_SIZE_K, GRID_SIZE_K);
		KernelDesityEstimation<<<dimGrid_K, BLOCK_SIZE>>>(dHs, dPoints, dAscii, dWeights);

		///////////////////////// STOP GPU EXCLUSIVE TIMING /////////////////////////////
	    // Record the stop event
	    error = hipEventRecord(stopExc, NULL);

	    if (error != hipSuccess)
	    {
	        printf("Failed to record stop event (error code %s)!\n", hipGetErrorString(error));
	        exit(EXIT_FAILURE);
	    }

	    // Wait for the stop event to complete
	    error = hipEventSynchronize(stopExc);
	    if (error != hipSuccess)
	    {
	        printf("Failed to synchronize on the stop event (error code %s)!\n", hipGetErrorString(error));
	        exit(EXIT_FAILURE);
	    }

	    elaps_exc = 0.0f;
	    error = hipEventElapsedTime(&elaps_exc, startExc, stopExc);

	    if (error != hipSuccess)
	    {
	        printf("Failed to get time elapsed between events (error code %s)!\n", hipGetErrorString(error));
	        exit(EXIT_FAILURE);
	    }
	    ///////////////////////// END OF STOP GPU EXCLUSIVE TIMING /////////////////////////////

		// copy results back to host
		CopyFromDeviceAsciiRaster(DenSurf_CUDA, dAscii);

		///////////////////////// STOP GPU INCLUSIVE TIMING /////////////////////////////
	    // Record the stop event
	    error = hipEventRecord(stopInc, NULL);

	    if (error != hipSuccess)
	    {
	        printf("Failed to record stop event (error code %s)!\n", hipGetErrorString(error));
	        exit(EXIT_FAILURE);
	    }

	    // Wait for the stop event to complete
	    error = hipEventSynchronize(stopInc);
	    if (error != hipSuccess)
	    {
	        printf("Failed to synchronize on the stop event (error code %s)!\n", hipGetErrorString(error));
	        exit(EXIT_FAILURE);
	    }

	    elaps_inc = 0.0f;
	    error = hipEventElapsedTime(&elaps_inc, startInc, stopInc);

	    if (error != hipSuccess)
	    {
	        printf("Failed to get time elapsed between events (error code %s)!\n", hipGetErrorString(error));
	        exit(EXIT_FAILURE);
	    }
	    ///////////////////////// END OF STOP GPU INCLUSIVE TIMING /////////////////////////////
	    printf("Computation on GPU took %.3f ms (EXCLUSIVE)\n", elaps_exc);
	    printf("Computation on GPU took %.3f ms (INCLUSIVE)\n", elaps_inc);

	    if(SKIPSEQ == 0){
			printf("SPEEDUP: %.3fx (EXCLUSIVE) %.3fx (INCLUSIVE)\n", elaps_seq / elaps_exc, elaps_seq / elaps_inc);
			// check resutls
			CheckResults(DenSurf, DenSurf_CUDA);
		}
		// write results to file
		WriteAsciiRaster(&DenSurf_CUDA, denCUDAfn);
		WriteSamplePoints(&Points, "pntsCUDA.csv");

		// clean up
		FreeDeviceSamplePoints(&dPoints);
		FreeDeviceEdgeCorrectionWeights(dWeights);
		FreeDeviceAsciiRaster(&dAscii);
		FreeSamplePoints(&Points);
		FreeAsciiRaster(&DenSurf);
		FreeAsciiRaster(&DenSurf_CUDA);
		FreeAsciiRaster(&Mask);
		FreeAsciiRaster(&dAscii);

		FreeBandwidths(hs);
		FreeDeviceBandwidths(dHs);
		FreeDen(den0);
		FreeDeviceDen(dDen0);
		FreeDeviceDen(dDen0cpy);
		FreeDen(den1);
		FreeDeviceDen(dDen1);
	}

	printf("Done...\n\n");

	return 0;
}

// mean center of points
void MeanCenter(SamplePoints Points, float &mean_x, float& mean_y){
	float sum_x = 0.0;
	float sum_y = 0.0;

	for (int p = 0; p < Points.numberOfPoints; p++){
		sum_x += Points.xCoordinates[p];
		sum_y += Points.yCoordinates[p];
	}

	mean_x = sum_x / Points.numberOfPoints;
	mean_y = sum_y / Points.numberOfPoints;
}

// standard distance squared
void StandardDistance2(SamplePoints Points, float &d2){

	float mean_x, mean_y;
	MeanCenter(Points, mean_x, mean_y);

	float sum2 = 0.0;

	for (int p = 0; p < Points.numberOfPoints; p++){
		sum2 += Distance2(mean_x, mean_y, Points.xCoordinates[p], Points.yCoordinates[p]);
	}

	d2 = sum2 / Points.numberOfPoints;
}

// generate random sample points
SamplePoints AllocateSamplePoints(int n){
	SamplePoints Points;

	Points.numberOfPoints = n;
	int size = n*sizeof(float);

	Points.xCoordinates = (float*)malloc(size);
	Points.yCoordinates = (float*)malloc(size);
	Points.weights = (float*)malloc(size);

	for (int i = 0; i < n; i++)
	{
		Points.xCoordinates[i] = rand() * 100.0f / RAND_MAX;
		Points.yCoordinates[i] = rand() * 100.0f / RAND_MAX;
		Points.weights[i] = 1.0f;
		//printf("x:%.2f y:%.2f w:%.2f\n", Points.xCoordinates[i], Points.yCoordinates[i], Points.weights[i]);
	}
	return Points;
}

// points read from a .csv file
SamplePoints ReadSamplePoints(const char *csvFile){
	FILE *f = fopen(csvFile, "rt");
	if (f == NULL)
	{
		printf("Error opening file!\n");
		exit(1);
	}

	const int CSV_LINE_LENGTH = 256;
	SamplePoints Points;
	int n = 0;
	char line[CSV_LINE_LENGTH];
	char ch;

	while (!feof(f))
	{
		ch = fgetc(f);
		if (ch == '\n')
		{
			n++;
		}
	}

	if (n == 1){
		printf("No point in file!\n");
		exit(1);
	}

	n = n - 1; // do not count the header line
	Points.numberOfPoints = n;
	Points.xCoordinates = (float*)malloc(n*sizeof(float));
	Points.yCoordinates = (float*)malloc(n*sizeof(float));
	Points.weights = (float*)malloc(n*sizeof(float));

	int counter = 0;
	char * pch;
	float x, y;
	rewind(f); // go back to the beginning of file
	fgets(line, CSV_LINE_LENGTH, f); //skip the header line
	while (fgets(line, CSV_LINE_LENGTH, f) != NULL){
		pch = strtok(line, ",\n");
		x = atof(pch);
		while (pch != NULL)
		{
			pch = strtok(NULL, ",\n");
			y = atof(pch);
			break;
		}
		Points.xCoordinates[counter] = x;
		Points.yCoordinates[counter] = y;
		Points.weights[counter] = 1.0;

		counter++;
	}

	fclose(f);

	return Points;
}

SamplePoints AllocateDeviceSamplePoints(const SamplePoints Points){
	SamplePoints dPoints = Points;
	dPoints.numberOfPoints = Points.numberOfPoints;
	int size = Points.numberOfPoints * sizeof(float);
	hipError_t error;
	error = hipMalloc((void**)&dPoints.xCoordinates, size);
	if (error != hipSuccess)
    {
        printf("ERROR in AllocateDeviceSamplePoints: %s\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
	error = hipMalloc((void**)&dPoints.yCoordinates, size);
	if (error != hipSuccess)
    {
        printf("ERROR in AllocateDeviceSamplePoints: %s\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
	error = hipMalloc((void**)&dPoints.weights, size);
	if (error != hipSuccess)
    {
        printf("ERROR in AllocateDeviceSamplePoints: %s\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
	return dPoints;
}

void CopyToDeviceSamplePoints(SamplePoints dPoints, const SamplePoints hPoints){
	int size = hPoints.numberOfPoints * sizeof(float);

	//for(int i = 0; i < hPoints.numberOfPoints; i++)
	//	printf("x:%.2f y:%.2f w:%.2f\n", hPoints.xCoordinates[i], hPoints.yCoordinates[i], hPoints.weights[i]);

	//printf("copy %d points to device\n", size);
	hipError_t error;

	error = hipMemcpy(dPoints.xCoordinates, hPoints.xCoordinates, size, hipMemcpyHostToDevice);
	if (error != hipSuccess)
    {
        printf("ERROR in CopyToDeviceSamplePoints: %s\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
	error = hipMemcpy(dPoints.yCoordinates, hPoints.yCoordinates, size, hipMemcpyHostToDevice);
		if (error != hipSuccess)
    {
        printf("ERROR in CopyToDeviceSamplePoints: %s\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
	error = hipMemcpy(dPoints.weights, hPoints.weights, size, hipMemcpyHostToDevice);
	if (error != hipSuccess)
    {
        printf("ERROR in CopyToDeviceSamplePoints: %s\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
}

// write to .csv file
void WriteSamplePoints(SamplePoints* Points, const char * csvFile){
	FILE *f = fopen(csvFile, "w");
	if (f == NULL)
	{
		printf("Error opening file!\n");
		exit(1);
	}

	fprintf(f, "x, y\n");
	for (int p = 0; p < Points->numberOfPoints; p++){
		fprintf(f, "%f, %f\n", Points->xCoordinates[p], Points->yCoordinates[p]);
	}
	fclose(f);
}

// write to .csv file
void WriteSamplePoints(SamplePoints* Points, float* Hs, float* Ws, const char * csvFile){
	FILE *f = fopen(csvFile, "w");
	if (f == NULL)
	{
		printf("Error opening file!\n");
		exit(1);
	}

	fprintf(f, "x, y, h, w\n");
	for (int p = 0; p < Points->numberOfPoints; p++){
		fprintf(f, "%f, %f, %f, %f\n", Points->xCoordinates[p], Points->yCoordinates[p], Hs[p], Ws[p]);
	}
	fclose(f);
}

void FreeSamplePoints(SamplePoints* Points){
	free(Points->xCoordinates);
	Points->xCoordinates = NULL;

	free(Points->yCoordinates);
	Points->yCoordinates = NULL;

	free(Points->weights);
	Points->weights = NULL;
}

void FreeDeviceSamplePoints(SamplePoints* dPoints){
	hipError_t error;
	error = hipFree(dPoints->xCoordinates);
	if (error != hipSuccess)
    {
        printf("ERROR in FreeDeviceSamplePoints: %s\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
	dPoints->xCoordinates = NULL;

	error = hipFree(dPoints->yCoordinates);
	if (error != hipSuccess)
    {
        printf("ERROR in FreeDeviceSamplePoints: %s\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
	dPoints->yCoordinates = NULL;

	error = hipFree(dPoints->weights);
	if (error != hipSuccess)
    {
        printf("ERROR in FreeDeviceSamplePoints: %s\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
	dPoints->weights = NULL;
}

// this is a mask
AsciiRaster AllocateAsciiRaster(int nCols, int nRows, float xLLCorner, float yLLCorner, float cellSize, float noDataValue){
	AsciiRaster Ascii;

	Ascii.nCols = nCols;
	Ascii.nRows = nRows;
	Ascii.xLLCorner = xLLCorner;
	Ascii.yLLCorner = yLLCorner;
	Ascii.cellSize = cellSize;
	Ascii.noDataValue = noDataValue;

	int size = Ascii.nCols * Ascii.nRows;
	Ascii.elements = (float*)malloc(size*sizeof(float));

	for (int row = 0; row < Ascii.nRows; row++){
		for (int col = 0; col < Ascii.nCols; col++){
			//if (row < 2 || col < 2)
			//	Ascii.elements[row * nCols + col] = Ascii.noDataValue;
			//else
				Ascii.elements[row * nCols + col] = 0.0f;
		}
	}

	return Ascii;
}

// copy a ascii raster
AsciiRaster CopyAsciiRaster(const AsciiRaster anotherAscii){
	AsciiRaster Ascii;

	Ascii.nCols = anotherAscii.nCols;
	Ascii.nRows = anotherAscii.nRows;
	Ascii.xLLCorner = anotherAscii.xLLCorner;
	Ascii.yLLCorner = anotherAscii.yLLCorner;
	Ascii.cellSize = anotherAscii.cellSize;
	Ascii.noDataValue = anotherAscii.noDataValue;

	int size = Ascii.nCols * Ascii.nRows;
	Ascii.elements = (float*)malloc(size*sizeof(float));

	for (int row = 0; row < Ascii.nRows; row++){
		for (int col = 0; col < Ascii.nCols; col++){
			Ascii.elements[row * Ascii.nCols + col] = anotherAscii.elements[row * Ascii.nCols + col];
		}
	}

	return Ascii;
}

// ascii raster read from a .asc file
AsciiRaster ReadAsciiRaster(char * asciiFile){
	FILE *f = fopen(asciiFile, "rt");
	if (f == NULL)
	{
		printf("Error opening file!\n");
		exit(1);
	}

	AsciiRaster Ascii;

	const int HEADER_LINE_LENGTH = 64;
	char hdrLine[HEADER_LINE_LENGTH];
	char* pch;
	float meta[6] = {0.f, 0.f, 0.f, 0.f, 0.f, 0.f};

	// read headers
	for (int i = 0; i < 6; i++){
		fgets(hdrLine, HEADER_LINE_LENGTH, f);
		pch = strtok(hdrLine, " \n");
		while (pch != NULL)
		{
			pch = strtok(NULL, "\n");
			meta[i] = atof(pch);
			break;
		}
	}

	Ascii.nCols = (int)meta[0];
	Ascii.nRows = (int)meta[1];
	Ascii.xLLCorner = meta[2];
	Ascii.yLLCorner = meta[3];
	Ascii.cellSize = meta[4];
	Ascii.noDataValue = meta[5];
	Ascii.elements = (float*)malloc(Ascii.nRows*Ascii.nCols*sizeof(float));

	const int DATA_LINE_LENGTH = Ascii.nCols * 32;
	char* datLine = (char*)malloc(DATA_LINE_LENGTH*sizeof(char));

	int row_counter = 0;
	while (fgets(datLine, DATA_LINE_LENGTH, f) != NULL){
		int col_counter = 0;
		pch = strtok(datLine, " \n");
		Ascii.elements[row_counter*Ascii.nCols+col_counter] = atof(pch);
		while (pch != NULL)
		{
			pch = strtok(NULL, " ");
			if (pch != NULL && col_counter < Ascii.nCols - 1){
				col_counter++;
				Ascii.elements[row_counter*Ascii.nCols + col_counter] = atof(pch);
			}
		}
		row_counter++;
	}
	free(datLine);

	fclose(f);

	return Ascii;
}

AsciiRaster AllocateDeviceAsciiRaster(const AsciiRaster hAscii){

	AsciiRaster dAscii = hAscii;
	dAscii.nCols = hAscii.nCols;
	dAscii.nRows = hAscii.nRows;
	dAscii.xLLCorner = hAscii.xLLCorner;
	dAscii.yLLCorner = hAscii.yLLCorner;
	dAscii.cellSize = hAscii.cellSize;
	dAscii.noDataValue = hAscii.noDataValue;

	int size = hAscii.nCols*hAscii.nRows * sizeof(float);
	hipError_t error;
	error = hipMalloc((void**)&dAscii.elements, size);
	if (error != hipSuccess)
    {
        printf("ERROR in AllocateDeviceAsciiRaster: %s\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
	return dAscii;
}

void CopyToDeviceAsciiRaster(AsciiRaster dAscii, const AsciiRaster hAscii){
	int size = hAscii.nCols*hAscii.nRows * sizeof(float);
	hipError_t error;
	error = hipMemcpy(dAscii.elements, hAscii.elements, size, hipMemcpyHostToDevice);
	if (error != hipSuccess)
    {
        printf("ERROR in CopyToDeviceAsciiRaster: %s\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
}

void CopyFromDeviceAsciiRaster(AsciiRaster hAscii, const AsciiRaster dAscii){
	hAscii.nCols = dAscii.nCols;
	hAscii.nRows = dAscii.nRows;
	hAscii.xLLCorner = dAscii.xLLCorner;
	hAscii.yLLCorner = dAscii.yLLCorner;
	hAscii.cellSize = dAscii.cellSize;
	hAscii.noDataValue = dAscii.noDataValue;

	int size = dAscii.nCols*dAscii.nRows * sizeof(float);
	hipError_t error;
	error = hipMemcpy(hAscii.elements, dAscii.elements, size, hipMemcpyDeviceToHost);
	if (error != hipSuccess)
    {
        printf("ERROR in CopyFromDeviceAsciiRaster: %s\n", hipGetErrorString(error));
		printf("size=%d mode=%d\n", size, hipMemcpyDeviceToHost);
        exit(EXIT_FAILURE);
    }
}

// write to .asc file
void WriteAsciiRaster(AsciiRaster* Ascii, const char * asciiFile){
	FILE *f = fopen(asciiFile, "w");
	if (f == NULL)
	{
		printf("Error opening file!\n");
		exit(1);
	}

	fprintf(f, "ncols %d\n", Ascii->nCols);
	fprintf(f, "nrows %d\n", Ascii->nRows);
	fprintf(f, "xllcorner %f\n", Ascii->xLLCorner);
	fprintf(f, "yllcorner %f\n", Ascii->yLLCorner);
	fprintf(f, "cellsize %f\n", Ascii->cellSize);
	fprintf(f, "NODATA_value %.0f\n", Ascii->noDataValue);

	for (int row = 0; row < Ascii->nRows; row++){
		for (int col = 0; col < Ascii->nCols; col++){
			fprintf(f, "%.16f ", Ascii->elements[row*Ascii->nCols+col]);
		}
		fprintf(f, "\n");
	}
	fclose(f);
}

void FreeAsciiRaster(AsciiRaster* Ascii){
	free(Ascii->elements);
	Ascii->elements = NULL;
}

void FreeDeviceAsciiRaster(AsciiRaster* Ascii){
	hipError_t error;
	error = hipFree(Ascii->elements);
	if (error != hipSuccess)
    {
        printf("ERROR in FreeDeviceAsciiRaster: %s\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
	Ascii->elements = NULL;
}

// edge effects correction weights at each point, weights is allocated somewhere else
void EdgeCorrectionWeightsExact(SamplePoints Points, float h, AsciiRaster Ascii, float *weights){
	float h2 = h * h;
	float cellArea = Ascii.cellSize * Ascii.cellSize;
	float p_x, p_y, cell_x, cell_y;
	float ew;

	for (int p = 0; p < Points.numberOfPoints; p++){
		//printf("%6d / %6d\n", p, Points.numberOfPoints);
		p_x = Points.xCoordinates[p];
		p_y = Points.yCoordinates[p];
		ew = 0.0f;
		for (int row = 0; row < Ascii.nRows; row++){
			for (int col = 0; col < Ascii.nCols; col++){
				if (Ascii.elements[row*Ascii.nCols+col] != Ascii.noDataValue){
					cell_x = COL_TO_XCOORD(col, Ascii.xLLCorner, Ascii.cellSize);
					cell_y = ROW_TO_YCOORD(row, Ascii.nRows, Ascii.yLLCorner, Ascii.cellSize);
					float d2 = Distance2(p_x, p_y, cell_x, cell_y);
					ew += GaussianKernel(h2, d2) * cellArea;
				}
			}
		}
		weights[p] = 1.0 / ew;
	}
}

void EdgeCorrectionWeightsExact(SamplePoints Points, float* hs, AsciiRaster Ascii, float *weights){
	//float h2 = BandWidth2(Points);
	float cellArea = Ascii.cellSize * Ascii.cellSize;
	float p_x, p_y, cell_x, cell_y;
	float ew, h2;

	for (int p = 0; p < Points.numberOfPoints; p++){
		//printf("%6d / %6d\n", p, Points.numberOfPoints);
		p_x = Points.xCoordinates[p];
		p_y = Points.yCoordinates[p];
		ew = 0.0f;
		h2 = hs[p] * hs[p];
		for (int row = 0; row < Ascii.nRows; row++){
			for (int col = 0; col < Ascii.nCols; col++){
				if (Ascii.elements[row*Ascii.nCols+col] != Ascii.noDataValue){
					cell_x = COL_TO_XCOORD(col, Ascii.xLLCorner, Ascii.cellSize);
					cell_y = ROW_TO_YCOORD(row, Ascii.nRows, Ascii.yLLCorner, Ascii.cellSize);
					float d2 = Distance2(p_x, p_y, cell_x, cell_y);
					ew += GaussianKernel(h2, d2) * cellArea;
				}
			}
		}
		weights[p] = 1.0 / ew;
	}
}

// approximate edge effects correction (Diggle 1985)
AsciiRaster EdgeCorrectionWeightsApprox(AsciiRaster Ascii, float h2){
	AsciiRaster weights = CopyAsciiRaster(Ascii);

	float curCell_x, curCell_y, iCell_x, iCell_y;
	float cellArea = Ascii.cellSize * Ascii.cellSize;

	for (int row = 0; row < Ascii.nRows; row++){
		curCell_y = ROW_TO_YCOORD(row, Ascii.nRows, Ascii.yLLCorner, Ascii.cellSize);
		for (int col = 0; col < Ascii.nCols; col++){
			curCell_x = COL_TO_XCOORD(col, Ascii.xLLCorner, Ascii.cellSize);
			int idx = row * Ascii.nCols + col;
			if (Ascii.elements[idx] != Ascii.noDataValue){
				float ew = 0.0f;
				for (int irow = 0; irow < Ascii.nRows; irow++){
					iCell_y = ROW_TO_YCOORD(irow, Ascii.nRows, Ascii.yLLCorner, Ascii.cellSize);
					for (int icol = 0; icol < Ascii.nCols; icol++){
						iCell_x = COL_TO_XCOORD(icol, Ascii.xLLCorner, Ascii.cellSize);
						float d2 = Distance2(curCell_x, curCell_y, iCell_x, iCell_y);
						ew += GaussianKernel(h2, d2) * cellArea;
					}
				}
				weights.elements[idx] = 1.0 / ew;
			}
		}
	}

	return weights;
}

float* AllocateEdgeCorrectionWeights(SamplePoints Points){
	return (float*)malloc(Points.numberOfPoints*sizeof(float));
}

float* AllocateDeviceEdgeCorrectionWeights(SamplePoints Points){
	float* dWeights;
	hipError_t error;
	error = hipMalloc((void**)&dWeights, Points.numberOfPoints*sizeof(float));
	if (error != hipSuccess)
    {
        printf("ERROR in AllocateDeviceEdgeCorrectionWeights: %s\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    return dWeights;
}

void FreeEdgeCorrectionWeights(float* weights){
	free(weights);
	weights = NULL;
}

void FreeDeviceEdgeCorrectionWeights(float* weights){
	hipError_t error;
	error = hipFree(weights);
	if (error != hipSuccess)
    {
        printf("ERROR in FreeDeviceEdgeCorrectionWeights: %s\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
	weights = NULL;
}

///////// Guiming on 2016-03-16 ///////////////
// the array holding bandwidth at each point
float* AllocateBandwidths(int n){ // n is number of points
	return (float*)malloc(n*sizeof(float));
}

float* AllocateDeviceBandwidths(int n){ // n is number of points
	float* dBandwidths;
	hipError_t error;
	error = hipMalloc((void**)&dBandwidths, n*sizeof(float));
	if (error != hipSuccess)
    {
        printf("ERROR in AllocateDeviceBandwidths: %s\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    return dBandwidths;
}

void CopyToDeviceBandwidths(float* dBandwidth, const float* hBandwidths, const int n){
	int size = n * sizeof(float);
	hipError_t error;
	error = hipMemcpy(dBandwidth, hBandwidths, size, hipMemcpyHostToDevice);
	if (error != hipSuccess)
    {
        printf("ERROR in CopyToDeviceBandwidths: %s\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
}

void CopyFromDeviceBandwidths(float* hBandwidth, const float* dBandwidths, const int n){
	int size = n * sizeof(float);
	hipError_t error;
	error = hipMemcpy(hBandwidth, dBandwidths, size, hipMemcpyDeviceToHost);
	if (error != hipSuccess)
    {
        printf("ERROR in CopyFromDeviceBandwidths: %s\n", hipGetErrorString(error));
		printf("size=%d mode=%d\n", size, hipMemcpyDeviceToHost);
        exit(EXIT_FAILURE);
    }
}

void FreeDeviceBandwidths(float* bandwidths){
	hipError_t error;
	error = hipFree(bandwidths);
	if (error != hipSuccess)
    {
        printf("ERROR in FreeDeviceBandwidths: %s\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
	bandwidths = NULL;
}

void FreeBandwidths(float* bandwidths){
	free(bandwidths);
	bandwidths = NULL;
}

// the array holding inclusive density at each point
float* AllocateDen(int n){ // n is number of points
	return (float*)malloc(n*sizeof(float));
}

float* AllocateDeviceDen(int n){ // n is number of points
	float* dDen;
	hipError_t error;
	error = hipMalloc((void**)&dDen, n*sizeof(float));
	if (error != hipSuccess)
    {
        printf("ERROR in AllocateDeviceDen: %s\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }

    return dDen;
}

void CopyToDeviceDen(float* dDen, const float* hDen, const int n){
	int size = n * sizeof(float);
	hipError_t error;
	error = hipMemcpy(dDen, hDen, size, hipMemcpyHostToDevice);
	if (error != hipSuccess)
    {
        printf("ERROR in CopyToDeviceDen: %s\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
}

void CopyFromDeviceDen(float* hDen, const float* dDen, const int n){
	int size = n * sizeof(float);
	hipError_t error;
	error = hipMemcpy(hDen, dDen, size, hipMemcpyDeviceToHost);
	if (error != hipSuccess)
    {
        printf("ERROR in CopyFromDeviceDen: %s\n", hipGetErrorString(error));
		printf("size=%d mode=%d\n", size, hipMemcpyDeviceToHost);
        exit(EXIT_FAILURE);
    }
}

void CopyDeviceDen(float* dDenTo, const float* dDenFrom, const int n){
	int size = n * sizeof(float);
	hipError_t error;
	error = hipMemcpy(dDenTo, dDenFrom, size, hipMemcpyDeviceToDevice);
	if (error != hipSuccess)
    {
        printf("ERROR in CopyDeviceDen: %s\n", hipGetErrorString(error));
		printf("size=%d mode=%d\n", size, hipMemcpyDeviceToDevice);
        exit(EXIT_FAILURE);
    }
}

void FreeDeviceDen(float* den){
	hipError_t error;
	error = hipFree(den);
	if (error != hipSuccess)
    {
        printf("ERROR in FreeDeviceDeviceDen: %s\n", hipGetErrorString(error));
        exit(EXIT_FAILURE);
    }
	den = NULL;
}

void FreeDen(float* den){
	free(den);
	den = NULL;
}

// compute the optimal Maximum Likelihood Estimation fixed bandwidth
// By Guiming @ 2016-02-26
float MLE_FixedBandWidth(AsciiRaster Ascii, SamplePoints Points, float *edgeWeights, float h, float* den0, float* den1, bool useGPU, float* dDen0, float* dDen1){

	float hA = h/10;
	float hD = 4 * h;
	float width = hD - hA;
	float epsilon = width/100;
	float factor = 1 + sqrtf(5.0f);
	int iteration = 0;
	while(width > epsilon){

		if(DEBUG){
			printf("iteration: %d ", iteration);
			printf("hD: %.6f ", hD);
			printf("hA: %.6f ", hA);
		}

		float hB = hA + width / factor;
		float hC = hD - width / factor;

		float LoghB = LogLikelihood(Ascii, Points, edgeWeights, hB, den0, den1, useGPU, dDen0, dDen1);
		float LoghC = LogLikelihood(Ascii, Points, edgeWeights, hC, den0, den1, useGPU, dDen0, dDen1);

		if(LoghB > LoghC){
			hD = hC;
			if(DEBUG) printf("LoghB: %.6f \n", LoghB);
		}
		else{
			hA = hB;
			if(DEBUG) printf("LoghC: %.6f \n", LoghC);
		}

		width = hD - hA;

		iteration += 1;
	}

	return (hA + hD) / 2;
}

// By Guiming @ 2016-05-21
// computed fixed bandwidth kde
void ComputeFixedDenistyAtPoints(AsciiRaster Ascii, SamplePoints Points, float *edgeWeights, float h, float* den0, float* den1, bool useGPU, float* dDen0, float* dDen1){
	int numPoints = Points.numberOfPoints;

	if(useGPU){ // do it on GPU
		// invoke kernels to compute density at each point
		// execution config.
		int NBLOCK_W = (numPoints + BLOCK_SIZE - 1) / BLOCK_SIZE;
	    int GRID_SIZE_W = (int)(sqrtf(NBLOCK_W)) + 1;
	    dim3 dimGrid_W(GRID_SIZE_W, GRID_SIZE_W);

	    // update edge correction weights
	    if(UPDATEWEIGHTS){
	    	CalcEdgeCorrectionWeights<<<dimGrid_W, BLOCK_SIZE>>>(h*h, Points, Ascii, edgeWeights);
	    }

		DensityAtPoints<<<dimGrid_W, BLOCK_SIZE>>>(h*h, Points, edgeWeights, dDen0, dDen1);
	}

	else{ // do it on CPU

		// update edge correction weights
		if(UPDATEWEIGHTS){
			EdgeCorrectionWeightsExact(Points, h, Ascii, edgeWeights);
		}

		for(int i = 0; i < numPoints; i++){
			float pi_x = Points.xCoordinates[i];
			float pi_y = Points.yCoordinates[i];

			float den = EPSILONDENSITY;
			float den_itself = EPSILONDENSITY;
			for (int j = 0; j < numPoints; j++){
				float pj_x = Points.xCoordinates[j];
				float pj_y = Points.yCoordinates[j];
				float pj_w = Points.weights[j];
				float pj_ew = edgeWeights[j];

				float d2 = Distance2(pi_x, pi_y, pj_x, pj_y);

				if(j == i){
					den_itself += GaussianKernel(h * h, d2) * pj_w *pj_ew; // / numPoints;
				}
				else{
					den += GaussianKernel(h * h, d2) * pj_w *pj_ew;
				}
			}

			if(den0 != NULL){
				den0[i] = den + den_itself;
			}
			if(den1 != NULL){
				den1[i] = den;
			}
		}
	}
}

// By Guiming @ 2016-02-26
// the log likelihood given single bandwidth h
float LogLikelihood(AsciiRaster Ascii, SamplePoints Points, float *edgeWeights, float h, float* den0, float* den1, bool useGPU, float* dDen0, float* dDen1){
	int numPoints = Points.numberOfPoints;
	float logL = 0.0f; // log likelihood

	if(useGPU){ // do it on GPU

		///*
		// execution config.
		int NBLOCK_W = (numPoints + BLOCK_SIZE - 1) / BLOCK_SIZE;
	    int GRID_SIZE_W = (int)(sqrtf(NBLOCK_W)) + 1;
	    dim3 dimGrid_W(GRID_SIZE_W, GRID_SIZE_W);

	    // update edge correction weights
	    if(UPDATEWEIGHTS){
	    	CalcEdgeCorrectionWeights<<<dimGrid_W, BLOCK_SIZE>>>(h*h, Points, Ascii, edgeWeights);
	    }

	    // invoke kernels to compute density at each point
		DensityAtPoints<<<dimGrid_W, BLOCK_SIZE>>>(h*h, Points, edgeWeights, dDen0, dDen1);
		//*/
		//ComputeFixedDenistyAtPoints(Ascii, Points, edgeWeights, h, NULL, NULL, true, NULL, dDen1);

		// compute likelihood on GPU
		ReductionSumGPU(dDen1, numPoints);
		hipMemcpyFromSymbol(&logL, HIP_SYMBOL(dReductionSum), sizeof(float), 0, hipMemcpyDeviceToHost);
		//printf("reduction result (likelihood) A: %3.4f \n", logL);
	}

	else{ // do it on CPU

		///*
		// update edge correction weights
		if(UPDATEWEIGHTS){
			EdgeCorrectionWeightsExact(Points, h, Ascii, edgeWeights);
		}

		for(int i = 0; i < numPoints; i++){
			float pi_x = Points.xCoordinates[i];
			float pi_y = Points.yCoordinates[i];

			float den = EPSILONDENSITY;
			float den_itself = EPSILONDENSITY;
			for (int j = 0; j < numPoints; j++){
				float pj_x = Points.xCoordinates[j];
				float pj_y = Points.yCoordinates[j];
				float pj_w = Points.weights[j];
				float pj_ew = edgeWeights[j];

				float d2 = Distance2(pi_x, pi_y, pj_x, pj_y);

				if(j == i){
					den_itself += GaussianKernel(h * h, d2) * pj_w *pj_ew; // / numPoints;
				}
				else{
					den += GaussianKernel(h * h, d2) * pj_w *pj_ew;
				}
			}

			logL = logL + log(den);

			if(den0 != NULL){
				den0[i] = den + den_itself;
			}
			if(den1 != NULL){
				den1[i] = den;
			}
		}//*/
		//ComputeFixedDenistyAtPoints(Ascii, Points, edgeWeights, h, NULL, den1, false, NULL, NULL);
		//for(int i = 0; i < numPoints; i++){
		//	logL = logL + log(den1[i]);
		//}
	}

	return logL;
}

// the log likelihood given bandwidths hs
// By Guiming @ 2016-02-26
// float* den0 : density based on all points, including itself
// float* den1 : leave one out density
float LogLikelihood(AsciiRaster Ascii, SamplePoints Points, float *edgeWeights, float* hs, float* den0, float* den1, bool useGPU, float* dHs, float* dDen0, float* dDen1, float h, float alpha, float* dDen0cpy){
	int numPoints = Points.numberOfPoints;
	float logL = 0.0f; // log likelihood

	if(useGPU){ // do it on GPU

		//CopyToDeviceBandwidths(dHs, hs, numPoints);

		// execution config.
		int NBLOCK_W = (numPoints + BLOCK_SIZE - 1) / BLOCK_SIZE;
	    int GRID_SIZE_W = (int)(sqrtf(NBLOCK_W)) + 1;
	    dim3 dimGrid_W(GRID_SIZE_W, GRID_SIZE_W);

	   	// update bandwidth on GPU
	    //CalcVaryingBandwidths<<<dimGrid_W, BLOCK_SIZE>>>(Points, dDen0, h, alpha, dHs);

	   	// update edge correction weights
	    if(UPDATEWEIGHTS){
	    	CalcEdgeCorrectionWeights<<<dimGrid_W, BLOCK_SIZE>>>(h*h, Points, Ascii, edgeWeights);
	    }

	    // compute (log) density at sample points [h^2, not h! OMG!!! Took me hours for spotting this!]
	    DensityAtPoints<<<dimGrid_W, BLOCK_SIZE>>>(h * h, Points, edgeWeights, dDen0, dDen1);

		// compute sum of log densities on GPU
		CopyDeviceDen(dDen0cpy, dDen0, numPoints);
		ReductionSumGPU(dDen0cpy, numPoints);
		//float tmp = 0.0f;
		//hipMemcpyFromSymbol(&tmp, HIP_SYMBOL(dReductionSum), sizeof(float), 0, hipMemcpyDeviceToHost);
		//printf("reduction result (geometricmean): %3.4f \n", exp(tmp/numPoints));


	    // update bandwidth on GPU
	    CalcVaryingBandwidths<<<dimGrid_W, BLOCK_SIZE>>>(Points, dDen0, h, alpha, dHs);

	    // update edge correction weights
	   	if(UPDATEWEIGHTS){
	    	CalcEdgeCorrectionWeights<<<dimGrid_W, BLOCK_SIZE>>>(dHs, Points, Ascii, edgeWeights);
	    }

		DensityAtPoints<<<dimGrid_W, BLOCK_SIZE>>>(dHs, Points, edgeWeights, dDen0, dDen1);

		// compute likelihood on GPU
		ReductionSumGPU(dDen1, numPoints);
		hipMemcpyFromSymbol(&logL, HIP_SYMBOL(dReductionSum), sizeof(float), 0, hipMemcpyDeviceToHost);
		//printf("reduction result (likelihood): %3.4f \n", logL);
	}
	else{ // do it on CPU

		// update edge correction weights
		if(UPDATEWEIGHTS){
			EdgeCorrectionWeightsExact(Points, h, Ascii, edgeWeights);
		}

		// compute den0 first
		for(int i = 0; i < numPoints; i++){
			float pi_x = Points.xCoordinates[i];
			float pi_y = Points.yCoordinates[i];

			float den = EPSILONDENSITY;
			for (int j = 0; j < numPoints; j++){
				float pj_x = Points.xCoordinates[j];
				float pj_y = Points.yCoordinates[j];
				float pj_w = Points.weights[j];
				float pj_ew = edgeWeights[j];

				float d2 = Distance2(pi_x, pi_y, pj_x, pj_y);
				den += GaussianKernel(h * h, d2) * pj_w *pj_ew;
			}

			if(den0 != NULL){
				den0[i] = den;
			}
		}

		// update bandwidths
		float gml = compGML(den0, numPoints);
		//printf("CPU reduction result (geometricmean): %3.4f \n", gml);
	    for(int i = 0; i < numPoints; i++){
	    	hs[i] = h * powf((den0[i] / gml), alpha);
	    }

		// update edge correction weights
		if(UPDATEWEIGHTS){
			EdgeCorrectionWeightsExact(Points, hs, Ascii, edgeWeights);
		}

		for(int i = 0; i < numPoints; i++){
			float pi_x = Points.xCoordinates[i];
			float pi_y = Points.yCoordinates[i];

			float den = EPSILONDENSITY;
			float den_itself = EPSILONDENSITY;
			for (int j = 0; j < numPoints; j++){
				float pj_x = Points.xCoordinates[j];
				float pj_y = Points.yCoordinates[j];
				float pj_h = hs[j];
				float pj_w = Points.weights[j];
				float pj_ew = edgeWeights[j];

				float d2 = Distance2(pi_x, pi_y, pj_x, pj_y);

				if(j == i){
					den_itself += GaussianKernel(pj_h * pj_h, d2) * pj_w *pj_ew; // / numPoints;
				}
				else{
					den += GaussianKernel(pj_h * pj_h, d2) * pj_w *pj_ew;
				}
			}

			logL = logL + log(den);

			if(den0 != NULL){
				den0[i] = den + den_itself;
			}
			if(den1 != NULL){
				den1[i] = den;
			}
		}
		//printf("CPU reduction result (likelihood): %3.4f \n", logL);
	}

	return logL;
}

// compute the log likelihood given a center (h0, alpha0) and step (stepH, stepA)
// By Guiming @ 2016-03-06
/*
 return 9 elements log likelihood in float* logLs
**/
void hj_likelihood(AsciiRaster Ascii, SamplePoints Points, float *edgeWeights, float h0, float alpha0, float stepH, float stepA, int lastdmax, float* logLs, float* hs, float* den0, float* den1, bool useGPU, float* dHs, float* dDen0, float* dDen1, float* dDen0cpy){

    //int n = Points.numberOfPoints;

    //float gml;

    // the center (h0, alpha0)
    if(lastdmax == -1){ // avoid unnecessary [expensive] computation
	    //LogLikelihood(Ascii, Points, edgeWeights, h0, den0, den1, useGPU, dDen0, dDen1);
	    float L0 = LogLikelihood(Ascii, Points, edgeWeights, hs, den0, den1, useGPU, dHs, dDen0, dDen1, h0, alpha0, dDen0cpy);
	    //printf("L0: %.5f\t", L0);
	    logLs[0] = L0;
	}

    // (h0 - stepH, alpha0)
    if(lastdmax != 2){ // avoid unnecessary [expensive] computation
	    //LogLikelihood(Ascii, Points, edgeWeights, h0 - stepH, den0, den1, useGPU, dDen0, dDen1);
	    float L1 = LogLikelihood(Ascii, Points, edgeWeights, hs, den0, den1, useGPU, dHs, dDen0, dDen1, h0 - stepH, alpha0, dDen0cpy);
	    //printf("L1: %.5f\t", L1);
	    logLs[1] = L1;
	}

    // (h0 + stepH, alpha0)
    if(lastdmax != 1){
	    //LogLikelihood(Ascii, Points, edgeWeights, h0 + stepH, den0, den1, useGPU, dDen0, dDen1);
	    float L2 = LogLikelihood(Ascii, Points, edgeWeights, hs, den0, den1, useGPU, dHs, dDen0, dDen1, h0 + stepH, alpha0, dDen0cpy);
	    //printf("L2: %.5f\t", L2);
	    logLs[2] = L2;
	}

    // (h0, alpha0 + stepA)
    if(lastdmax != 4){
	    //LogLikelihood(Ascii, Points, edgeWeights, h0, den0, den1, useGPU, dDen0, dDen1);
	    float L3 = LogLikelihood(Ascii, Points, edgeWeights, hs, den0, den1, useGPU, dHs, dDen0, dDen1, h0, alpha0 + stepA, dDen0cpy);
	    //printf("L3: %.5f\t", L3);
	    logLs[3] = L3;
	}

    // (h0, alpha0 - stepA)
    if(lastdmax != 3){
	    //LogLikelihood(Ascii, Points, edgeWeights, h0, den0, den1, useGPU, dDen0, dDen1);
	    float L4 = LogLikelihood(Ascii, Points, edgeWeights, hs, den0, den1, useGPU, dHs, dDen0, dDen1, h0, alpha0 - stepA, dDen0cpy);
	    //printf("L4: %.5f\n", L4);
	    logLs[4] = L4;
	}
}

// compute the optimal h and alpha (parameters for calculating the optimal adaptive bandwith)
// By Guiming @ 2016-03-06
/*
 return 3 optmal parameters in float* optParas (optH, optAlpha, LogLmax)
**/
void hooke_jeeves(AsciiRaster Ascii, SamplePoints Points, float *edgeWeights, float h0, float alpha0, float stepH, float stepA, float* optParas, float* hs, float* den0, float* den1, bool useGPU, float* dHs, float* dDen0, float* dDen1, float* dDen0cpy){
	float* Ls = (float*)malloc(5 * sizeof(float)); // remeber to free at the end
	hj_likelihood(Ascii, Points, edgeWeights, h0, alpha0, stepH, stepA, -1, Ls, hs, den0, den1, useGPU, dHs, dDen0, dDen1, dDen0cpy);

	float Lmax = Ls[0];

	float s = stepH / 20;
	float a = stepA / 20;

	int iteration = 0;
    while ((stepH > s || stepA > a) &&  iteration <= MAX_NUM_ITERATIONS){

        //float Lmax0 = Lmax;
        int dmax = 0;
        for(int i = 0; i < 5; i++){
            if(Ls[i] > Lmax){
            	Lmax = Ls[i];
                dmax = i;
            }
        }
        if(DEBUG)
        	printf ("iteration: %d center: (%.5f %.5f) steps: (%.5f %.5f) dmax: %d Lmax: %.5f \n", iteration, h0, alpha0, stepH, stepA, dmax, Lmax);

        if(dmax == 0){
            stepH = stepH / 2;
            stepA = stepA / 2;
        }

        else{
            if(dmax == 1){
                h0 = h0 - stepH;
                alpha0 = alpha0;
                Ls[2] = Ls[0]; // avoid unnecessary [expensive] computation
                Ls[0] = Ls[1];
            }
            if(dmax == 2){
                h0 = h0 + stepH;
                alpha0 = alpha0;
                Ls[1] = Ls[0];
                Ls[0] = Ls[2];
            }
            if (dmax == 3){
                h0 = h0;
                alpha0 = alpha0 + stepA;
                Ls[3] = Ls[0];
                Ls[0] = Ls[4];
            }
            if(dmax == 4){
                h0 = h0;
                alpha0 = alpha0 - stepA;
                Ls[3] = Ls[0];
                Ls[0] = Ls[4];
            }
        }
	    hj_likelihood(Ascii, Points, edgeWeights, h0, alpha0, stepH, stepA, dmax, Ls, hs, den0, den1, useGPU, dHs, dDen0, dDen1, dDen0cpy);

	    iteration++;
    }

    optParas[0] = h0;
    optParas[1] = alpha0;
    optParas[2] = Lmax;

    free(Ls);
    Ls = NULL;
}

///////// Guiming on 2016-03-16 ///////////////

// check whether the result from sequential computation and that from parallel computation agree
void CheckResults(AsciiRaster AsciiSEQ, AsciiRaster AsciiPARA){
	float eps = 0.000001f;

	int n = AsciiSEQ.nCols * AsciiSEQ.nRows;

	for(int i = 0; i < n; i++){
		if(abs(AsciiSEQ.elements[i] - AsciiPARA.elements[i]) > eps){
			printf("TEST FAILED. Result from parallel computation does not match that from sequential computation.\n");
			return;
		}
	}
	printf("TEST PASSED. Result from GPU computation does match that from CPU computation.\n");
}

float compGML(float* den0, int n){
	float gml = 0.0f;
	for(int i = 0; i < n; i++){
		gml = gml + log(den0[i]);
	}
	gml = expf(gml / n);
	return gml;
}

// reduction sum on GPU
void ReductionSumGPU(float* dArray, int numberOfElements){

   unsigned int N = numberOfElements;

   int iteration = 0;
   int NUM_ACTIVE_ITEMS = numberOfElements; // # active items need to be reduced

   // approx. # of blocks needed
   int NUM_BLOCKS = (numberOfElements ) / BLOCK_SIZE;

   // decide grid dimension
   int GRID_SIZE = (int)(sqrtf(NUM_BLOCKS)) + 1;
   dim3 dimGrid(GRID_SIZE, GRID_SIZE);

   // call the kernel for the first iteration
   ReductionSum<<<dimGrid, BLOCK_SIZE>>>(dArray, N, iteration, NUM_ACTIVE_ITEMS);

   // update # of items to be reduced in next iteration
   NUM_ACTIVE_ITEMS = (NUM_ACTIVE_ITEMS + BLOCK_SIZE - 1) / BLOCK_SIZE;

   // update numberOfElements (needed for deciding grid dimension)
   numberOfElements = dimGrid.x * dimGrid.y;

   // increment iteraton index
   iteration++;

   // iterate if needed
   while(numberOfElements > 1){
      NUM_BLOCKS = (numberOfElements ) / BLOCK_SIZE;

      GRID_SIZE = (int)(sqrtf(NUM_BLOCKS)) + 1;
      dimGrid.x = GRID_SIZE;
      dimGrid.y = GRID_SIZE;
      ReductionSum<<<dimGrid, BLOCK_SIZE>>>(dArray, N, iteration, NUM_ACTIVE_ITEMS);
      NUM_ACTIVE_ITEMS = (NUM_ACTIVE_ITEMS + BLOCK_SIZE - 1) / BLOCK_SIZE;

      numberOfElements = dimGrid.x * dimGrid.y;

      iteration++;
   }
}
