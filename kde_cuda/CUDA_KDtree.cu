#include "hip/hip_runtime.h"
#include "CUDA_KDtree.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <float.h>
#include <cstdio>

void CheckCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }
}

/*
__device__ float Distance(const Point &a, const Point &b)
{
    float deltaX = a.coords[0] - b.coords[0];
    float deltaY = a.coords[1] - b.coords[1];
    return deltaX * deltaX + deltaY * deltaY;
}
*/

/*
__device__ void SearchAtNode(const CUDA_KDNode *nodes, const int *indexes, const Point *pts, int cur, const Point &query, int *ret_index, float *ret_dist, int *ret_node)
{
    // Finds the first potential candidate

    int best_idx = 0;
    float best_dist = FLT_MAX;

    while(true) {
        int split_axis = nodes[cur].level % KDTREE_DIM;

        if(nodes[cur].left == -1) {
            *ret_node = cur;

            for(int i=0; i < nodes[cur].num_indexes; i++) {
                int idx = indexes[nodes[cur].indexes + i];
                float dist = Distance(query, pts[idx]);
                if(dist < best_dist) {
                    best_dist = dist;
                    best_idx = idx;
                }
            }

            break;
        }
        else if(query.coords[split_axis] < nodes[cur].split_value) {
            cur = nodes[cur].left;
        }
        else {
            cur = nodes[cur].right;
        }
    }

    *ret_index = best_idx;
    *ret_dist = best_dist;
}

__device__ void SearchAtNodeRange(const CUDA_KDNode *nodes, const int *indexes, const Point *pts, const Point &query,int cur, float range, int *ret_index, float *ret_dist)
{
    // Goes through all the nodes that are within "range"

    int best_idx = 0;
    float best_dist = FLT_MAX;

    // Ok, we don't have nice STL vectors to use, and we can't dynamically allocate memory with CUDA??
    // We'll use a fixed length stack, increase this as required
    int to_visit[CUDA_STACK];
    int to_visit_pos = 0;

    to_visit[to_visit_pos++] = cur;

    while(to_visit_pos) {
        int next_search[CUDA_STACK];
        int next_search_pos = 0;

        while(to_visit_pos) {
            cur = to_visit[to_visit_pos-1];
            to_visit_pos--;

            int split_axis = nodes[cur].level % KDTREE_DIM;

            if(nodes[cur].left == -1) {
                for(int i=0; i < nodes[cur].num_indexes; i++) {
                    int idx = indexes[nodes[cur].indexes + i];
                    float d = Distance(query, pts[idx]);

                    if(d < best_dist) {
                        best_dist = d;
                        best_idx = idx;
                    }
                }
            }
            else {
                float d = query.coords[split_axis] - nodes[cur].split_value;

                // There are 3 possible scenarios
                // The hypercircle only intersects the left region
                // The hypercircle only intersects the right region
                // The hypercricle intersects both

                if(fabs(d) > range) {
                    if(d < 0)
                        next_search[next_search_pos++] = nodes[cur].left;
                    else
                        next_search[next_search_pos++] = nodes[cur].right;
                }
                else {
                    next_search[next_search_pos++] = nodes[cur].left;
                    next_search[next_search_pos++] = nodes[cur].right;
                }
            }
        }

        // No memcpy available??
        for(int i=0; i  < next_search_pos; i++)
            to_visit[i] = next_search[i];

        to_visit_pos = next_search_pos;
    }

    *ret_index = best_idx;
    *ret_dist = best_dist;
}


__device__ void Search(const CUDA_KDNode *nodes, const int *indexes, const Point *pts, const Point &query, int *ret_index, float *ret_dist)
{
    // Find the first closest node, this will be the upper bound for the next searches
    int best_node = 0;
    int best_idx = 0;
    float best_dist = FLT_MAX;
    float radius = 0;

    SearchAtNode(nodes, indexes, pts, 0, query, &best_idx, &best_dist, &best_node);

    radius = sqrt(best_dist);

    // Now find other possible candidates
    int cur = best_node;

    while(nodes[cur].parent != -1) {
        // Go up
        int parent = nodes[cur].parent;
        int split_axis = nodes[parent].level % KDTREE_DIM;

        // Search the other node
        float tmp_dist = FLT_MAX;
        int tmp_idx;

        if(fabs(nodes[parent].split_value - query.coords[split_axis]) <= radius) {
            // Search opposite node
            if(nodes[parent].left != cur)
                SearchAtNodeRange(nodes, indexes, pts, query, nodes[parent].left, radius, &tmp_idx, &tmp_dist);
            else
                SearchAtNodeRange(nodes, indexes, pts, query, nodes[parent].right, radius, &tmp_idx, &tmp_dist);
        }

        if(tmp_dist < best_dist) {
            best_dist = tmp_dist;
            best_idx = tmp_idx;
        }

        cur = parent;
    }

    *ret_index = best_idx;
    *ret_dist = best_dist;
}
*/
/*
__global__ void dSearchRange(const CUDA_KDNode *nodes, const int *indexes, const Point *pts, const Point &query, const float range, int &ret_num_nbrs, int *ret_indexes, float *ret_dists)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if(idx >= 1)
        return;

    //printf("GPU: x=%f y=%f z=%f\n", query.coords[0], query.coords[1], query.coords[2]);

    // Goes through all the nodes that are within "range"
    int cur = 0; // root
    int num_nbrs = 0;

    // Ok, we don't have nice STL vectors to use, and we can't dynamically allocate memory with CUDA??
    // We'll use a fixed length stack, increase this as required
    int to_visit[CUDA_STACK];
    int to_visit_pos = 0;

    to_visit[to_visit_pos++] = cur;

    while(to_visit_pos) {
        int next_search[CUDA_STACK];
        int next_search_pos = 0;

        while(to_visit_pos) {
            cur = to_visit[to_visit_pos-1];
            to_visit_pos--;

            int split_axis = nodes[cur].level % KDTREE_DIM;

            if(nodes[cur].left == -1) {
                for(int i=0; i < nodes[cur].num_indexes; i++) {
                    int idx = indexes[nodes[cur].indexes + i];
                    float d = Distance(query, pts[idx]);

                    if(d < range) {
                        ret_indexes[num_nbrs] = idx;
                        ret_dists[num_nbrs] = d;
                        num_nbrs++;
                    }
                }
            }
            else {
                float d = query.coords[split_axis] - nodes[cur].split_value;

                // There are 3 possible scenarios
                // The hypercircle only intersects the left region
                // The hypercircle only intersects the right region
                // The hypercricle intersects both

                if(fabs(d*d) > range) {
                    if(d < 0)
                        next_search[next_search_pos++] = nodes[cur].left;
                    else
                        next_search[next_search_pos++] = nodes[cur].right;
                }
                else {
                    next_search[next_search_pos++] = nodes[cur].left;
                    next_search[next_search_pos++] = nodes[cur].right;
                }
            }
        }

        // No memcpy available??
        for(int i=0; i  < next_search_pos; i++)
            to_visit[i] = next_search[i];

        to_visit_pos = next_search_pos;
    }
    //printf("A:ret_num_nbrs=%d\n", num_nbrs);
    ret_num_nbrs = num_nbrs;
    //printf("B:ret_num_nbrs=%d\n", ret_num_nbrs);
}*/

/*
__global__ void SearchBatch(const CUDA_KDNode *nodes, const int *indexes, const Point *pts, int num_pts, Point *queries, int num_queries, int *ret_index, float *ret_dist)
{
    int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if(idx >= num_queries)
        return;

    Search(nodes, indexes, pts, queries[idx], &ret_index[idx], &ret_dist[idx]);
}
*/
//CUDA_KDTree::CUDA_KDTree()
//{
//}

CUDA_KDTree::~CUDA_KDTree()
{
    hipFree(m_gpu_nodes);
    hipFree(m_gpu_indexes);
    hipFree(m_gpu_points);
}

void CUDA_KDTree::CreateKDTree(KDNode *root, int num_nodes, const vector <Point> &data)
{
    // Create the nodes again on the CPU, laid out nicely for the GPU transfer
    // Not exactly memory efficient, since we're creating the entire tree again
    m_num_points = data.size();

    hipMalloc((void**)&m_gpu_nodes, sizeof(CUDA_KDNode)*num_nodes);
    hipMalloc((void**)&m_gpu_indexes, sizeof(int)*m_num_points);
    hipMalloc((void**)&m_gpu_points, sizeof(Point)*m_num_points);

    CheckCUDAError("CreateKDTree");

    vector <CUDA_KDNode> cpu_nodes(num_nodes);
    vector <int> indexes(m_num_points);
    vector <KDNode*> to_visit;

    int cur_pos = 0;

    to_visit.push_back(root);

    while(to_visit.size()) {
        vector <KDNode*> next_search;

        while(to_visit.size()) {
            KDNode *cur = to_visit.back();
            to_visit.pop_back();

            int id = cur->id;

            cpu_nodes[id].level = cur->level;
            cpu_nodes[id].parent = cur->_parent;
            cpu_nodes[id].left = cur->_left;
            cpu_nodes[id].right = cur->_right;
            cpu_nodes[id].split_value = cur->split_value;
            cpu_nodes[id].num_indexes = cur->indexes.size();

            if(cur->indexes.size()) {
                for(unsigned int i=0; i < cur->indexes.size(); i++)
                    indexes[cur_pos+i] = cur->indexes[i];

                cpu_nodes[id].indexes = cur_pos;
                cur_pos += cur->indexes.size();
            }
            else {
                cpu_nodes[id].indexes = -1;
            }

            if(cur->left)
                next_search.push_back(cur->left);

            if(cur->right)
                next_search.push_back(cur->right);
        }

        to_visit = next_search;
    }

    hipMemcpy(m_gpu_nodes, &cpu_nodes[0], sizeof(CUDA_KDNode)*cpu_nodes.size(), hipMemcpyHostToDevice);
    hipMemcpy(m_gpu_indexes, &indexes[0], sizeof(int)*indexes.size(), hipMemcpyHostToDevice);
    hipMemcpy(m_gpu_points, &data[0], sizeof(Point)*data.size(), hipMemcpyHostToDevice);

    CheckCUDAError("CreateKDTree");
}
/*
void CUDA_KDTree::Search(const vector <Point> &queries, vector <int> &indexes, vector <float> &dists)
{
    int threads = 512;
    int blocks = queries.size()/threads + ((queries.size() % threads)?1:0);

    Point *gpu_queries;
    int *gpu_ret_indexes;
    float *gpu_ret_dist;

    indexes.resize(queries.size());
    dists.resize(queries.size());

    hipMalloc((void**)&gpu_queries, sizeof(Point)*queries.size()*KDTREE_DIM);
    hipMalloc((void**)&gpu_ret_indexes, sizeof(int)*queries.size()*KDTREE_DIM);
    hipMalloc((void**)&gpu_ret_dist, sizeof(float)*queries.size()*KDTREE_DIM);

    CheckCUDAError("Search");

    hipMemcpy(gpu_queries, &queries[0], sizeof(float)*queries.size()*KDTREE_DIM, hipMemcpyHostToDevice);

    CheckCUDAError("Search");

    printf("CUDA blocks/threads: %d %d\n", blocks, threads);

    SearchBatch<<<blocks, threads>>>(m_gpu_nodes, m_gpu_indexes, m_gpu_points, m_num_points, gpu_queries, queries.size(), gpu_ret_indexes, gpu_ret_dist);
    hipDeviceSynchronize();

    CheckCUDAError("Search");

    hipMemcpy(&indexes[0], gpu_ret_indexes, sizeof(int)*queries.size(), hipMemcpyDeviceToHost);
    hipMemcpy(&dists[0], gpu_ret_dist, sizeof(float)*queries.size(), hipMemcpyDeviceToHost);

    hipFree(gpu_queries);
    hipFree(gpu_ret_indexes);
    hipFree(gpu_ret_dist);
}
*/

/*
void CUDA_KDTree::SearchRange(const Point &query, const float range, vector <int> &indexes, vector <float> &dists)
{
    int threads = 512;
    int blocks = 1;

    int num_nbrs;
    int *gpu_num_nbrs;

    Point *gpu_query;
    int *gpu_ret_indexes;
    float *gpu_ret_dist;

    int NPNTS = GetNumPoints();

    hipMalloc((void**)&gpu_num_nbrs, sizeof(int));
    hipMalloc((void**)&gpu_query, sizeof(Point)*KDTREE_DIM);
    hipMalloc((void**)&gpu_ret_indexes, sizeof(int)*NPNTS*KDTREE_DIM);
    hipMalloc((void**)&gpu_ret_dist, sizeof(float)*NPNTS*KDTREE_DIM);

    CheckCUDAError("Search");

    hipMemcpy(gpu_query, &query, sizeof(float)*KDTREE_DIM, hipMemcpyHostToDevice);

    CheckCUDAError("Search");

    //printf("CUDA blocks/threads: %d %d\n", blocks, threads);
    //SearchRange(const CUDA_KDNode *nodes, const int *indexes, const Point *pts, const Point &query, const float range, int &ret_num_nbrs, int *ret_indexes, float *ret_dists)
    dSearchRange<<<blocks, threads>>>(m_gpu_nodes, m_gpu_indexes, m_gpu_points, *gpu_query, range, *gpu_num_nbrs, gpu_ret_indexes, gpu_ret_dist);
    hipDeviceSynchronize();

    hipMemcpy(&num_nbrs, gpu_num_nbrs, sizeof(int), hipMemcpyDeviceToHost);
    //printf("num_nbrs=%d\n", num_nbrs);
    CheckCUDAError("Search");

    indexes.resize(num_nbrs);
    dists.resize(num_nbrs);

    hipMemcpy(&indexes[0], gpu_ret_indexes, sizeof(int)*num_nbrs, hipMemcpyDeviceToHost);
    hipMemcpy(&dists[0], gpu_ret_dist, sizeof(float)*num_nbrs, hipMemcpyDeviceToHost);

    hipFree(gpu_query);
    hipFree(gpu_ret_indexes);
    hipFree(gpu_ret_dist);
}
*/
