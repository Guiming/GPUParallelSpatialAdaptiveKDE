#include "hip/hip_runtime.h"
// Copyright 2016 Guiming Zhang (gzhang45@wisc.edu)
// Distributed under GNU General Public License (GPL) license

/*
* Kernel function
*/

#ifndef _KDE_KERNEL_H_
#define _KDE_KERNEL_H_

#include <stdio.h>
#include <stdlib.h>
#include <math_functions.h>

#include "SamplePoints.h"
#include "AsciiRaster.h"
#include "Utilities.h"

__device__ float dReductionSum = 1.0f; // sum of log of densities
__device__ float dDen0_0 = 1.0f; // sum of log of densities

// squared distance btw two points
__device__  float dDistance2(float x0, float y0, float x1, float y1){
	float dx = x1 - x0;
	float dy = y1 - y0;
	return dx*dx + dy*dy;
}

// Gaussian kernel
__device__ float dGaussianKernel(float h2, float d2){
	return expf(d2 / (-2.0f * h2)) / (h2 * TWO_PI);
}

// Edge correction with fixed bandwidth h2 (squared)
__global__ void CalcEdgeCorrectionWeights(float h2, const SamplePoints dPoints, const AsciiRaster dAscii, float* dWeights)
{

	// serial point ID
	int tid = (blockIdx.y * gridDim.y + blockIdx.x) * blockDim.x + threadIdx.x;

	// directly return if ID goes out of range
	if(tid >= dPoints.numberOfPoints){
		return;
	}

	// otherwise calculate edge effect correction weight point ID = tid
	float cellSize = dAscii.cellSize;
	int nCols = dAscii.nCols;
	int nRows = dAscii.nRows;
	float xLLCorner = dAscii.xLLCorner;
	float yLLCorner = dAscii.yLLCorner;
	float noDataValue = dAscii.noDataValue;

	//printf("%d %d\n", nCols, nRows);

	float cellArea = cellSize * cellSize;

	float p_x = dPoints.xCoordinates[tid];
	float p_y = dPoints.yCoordinates[tid];
	float ew = 0.0f;

	//printf("%d %.3f %.3f\n", tid, p_x, p_y);

	float cell_x, cell_y, val, d2;//, g;
	int row, col;
	for (row = 0; row < nRows; row++){
		for (col = 0; col < nCols; col++){
			val = dAscii.elements[row*nCols+col];
			if (val != noDataValue){
				cell_x = COL_TO_XCOORD(col, xLLCorner, cellSize);
				cell_y = ROW_TO_YCOORD(row, nRows, yLLCorner, cellSize);
				d2 = dDistance2(p_x, p_y, cell_x, cell_y);
				//if(d2 < 9.0 * h2){
					ew += dGaussianKernel(h2, d2) * cellArea;
				//}
			}
		}
	}
	dWeights[tid] = 1.0f / ew;
}

// Edge correction with adaptive bandwidth (variable bandwidth at each point in dHs)
__global__ void CalcEdgeCorrectionWeights(float* dHs, const SamplePoints dPoints, const AsciiRaster dAscii, float* dWeights)
{

	// serial point ID
	int tid = (blockIdx.y * gridDim.y + blockIdx.x) * blockDim.x + threadIdx.x;

	// directly return if ID goes out of range
	if(tid >= dPoints.numberOfPoints){
		return;
	}

	// otherwise calculate edge effect correction weight point ID = tid
	float cellSize = dAscii.cellSize;
	int nCols = dAscii.nCols;
	int nRows = dAscii.nRows;
	float xLLCorner = dAscii.xLLCorner;
	float yLLCorner = dAscii.yLLCorner;
	float noDataValue = dAscii.noDataValue;

	//printf("%d %d\n", nCols, nRows);

	float cellArea = cellSize * cellSize;

	float p_x = dPoints.xCoordinates[tid];
	float p_y = dPoints.yCoordinates[tid];
	float ew = 0.0f;
	float h = dHs[tid];
	float h2 = h * h;

	//printf("%d %.3f %.3f\n", tid, p_x, p_y);

	float cell_x, cell_y, val, d2;//, g;
	int row, col;
	for (row = 0; row < nRows; row++){
		for (col = 0; col < nCols; col++){
			val = dAscii.elements[row*nCols+col];
			if (val != noDataValue){
				cell_x = COL_TO_XCOORD(col, xLLCorner, cellSize);
				cell_y = ROW_TO_YCOORD(row, nRows, yLLCorner, cellSize);
				d2 = dDistance2(p_x, p_y, cell_x, cell_y);

				//if(d2 < 9.0 * h2){
					ew += dGaussianKernel(h2, d2) * cellArea;
				//}
			}
		}
	}
	dWeights[tid] = 1.0f / ew;
}


// Kernel density estimation with fixed bandwidth h2 (squared)
__global__ void KernelDesityEstimation(float h2, const SamplePoints dPoints, const AsciiRaster dAscii, float* dWeights)
{
	// serial point ID
	int tid = (blockIdx.y * gridDim.y + blockIdx.x) * blockDim.x + threadIdx.x;

	// # of rows and cols
	int nCols = dAscii.nCols;
	int nRows = dAscii.nRows;

	// directly return if ID goes out of range
	if(tid >= nCols * nRows){
		return;
	}

	// otherwise, do KDE
	float cellSize = dAscii.cellSize;
	float xLLCorner = dAscii.xLLCorner;
	float yLLCorner = dAscii.yLLCorner;
	float noDataValue = dAscii.noDataValue;
	float cell_x, cell_y; // x,y coord of cell
	float p_x, p_y, p_w;    // x, y coord, weight of point
	int numPoints = dPoints.numberOfPoints;
	float d2;
	float e_w = 1.0;    // edge effect correction weight
	float den;
	int col, row;

	// which row, col?
	row = tid / nCols;
	col = tid - row * nCols;

	// x, y coord of this cell
	cell_y = ROW_TO_YCOORD(row, nRows, yLLCorner, cellSize);
	cell_x = COL_TO_XCOORD(col, xLLCorner, cellSize);

	// should do KDE on this cell?
	float val = dAscii.elements[tid];

	if(val == noDataValue) {
		return;
	}

	den = 0.0f;
	for (int p = 0; p < numPoints; p++){
		p_x = dPoints.xCoordinates[p];
		p_y = dPoints.yCoordinates[p];
		p_w = dPoints.weights[p];
		e_w = dWeights[p];
		d2 = dDistance2(p_x, p_y, cell_x, cell_y);

		//if(d2 < 9.0 * h2){
			den += dGaussianKernel(h2, d2) * p_w *e_w;
		//}
	}
	dAscii.elements[tid] = den; // intensity, not probability
}

// Kernel density estimation with adaptive bandwidth (variable bandwidth at each point in dHs)
__global__ void KernelDesityEstimation(float* dHs, const SamplePoints dPoints, const AsciiRaster dAscii, float* dWeights)
{
	// serial point ID
	int tid = (blockIdx.y * gridDim.y + blockIdx.x) * blockDim.x + threadIdx.x;

	// # of rows and cols
	int nCols = dAscii.nCols;
	int nRows = dAscii.nRows;

	// directly return if ID goes out of range
	if(tid >= nCols * nRows){
		return;
	}

	// otherwise, do KDE
	float cellSize = dAscii.cellSize;
	float xLLCorner = dAscii.xLLCorner;
	float yLLCorner = dAscii.yLLCorner;
	float noDataValue = dAscii.noDataValue;
	float cell_x, cell_y; // x,y coord of cell
	float p_x, p_y, p_w;    // x, y coord, weight of point
	int numPoints = dPoints.numberOfPoints;
	float h, d2;
	float e_w = 1.0f;    // edge effect correction weight
	float den;
	int col, row;

	// which row, col?
	row = tid / nCols;
	col = tid - row * nCols;

	// x, y coord of this cell
	cell_y = ROW_TO_YCOORD(row, nRows, yLLCorner, cellSize);
	cell_x = COL_TO_XCOORD(col, xLLCorner, cellSize);

	// should do KDE on this cell?
	float val = dAscii.elements[tid];

	if(val == noDataValue) {
		return;
	}

	den = 0.0f;
	for (int p = 0; p < numPoints; p++){
		p_x = dPoints.xCoordinates[p];
		p_y = dPoints.yCoordinates[p];
		p_w = dPoints.weights[p];
		e_w = dWeights[p];
		h = dHs[p];
		d2 = dDistance2(p_x, p_y, cell_x, cell_y);

		//if(d2 < 9.0 * h * h){
			den += dGaussianKernel(h * h, d2) * p_w *e_w;
		//}

		//den += dGaussianKernel(h * h, d2) * p_w *e_w;
	}
	dAscii.elements[tid] = den; // intensity, not probability
}

// Density at each point under fixed bandwidth h2 (squared)
__global__ void DensityAtPoints(float h2, const SamplePoints dPoints, float *dWeights, float* dDen0, float* dDen1){
	// serial point ID
	int tid = (blockIdx.y * gridDim.y + blockIdx.x) * blockDim.x + threadIdx.x;

	int n = dPoints.numberOfPoints;
	// directly return if ID goes out of range
	if(tid >= n){
		return;
	}

	// otherwise calculate density at point ID = tid
	float p_x = dPoints.xCoordinates[tid];
	float p_y = dPoints.yCoordinates[tid];
	float den = 0.0f;

	//printf("%d %.3f %.3f\n", tid, p_x, p_y);
	int i;
	float x, y, p_w, e_w, d2;//, g;
	float den_itselft = 0.0f;
	for (i = 0; i < n; i++){
		x = dPoints.xCoordinates[i];
		y = dPoints.yCoordinates[i];
		p_w = dPoints.weights[i];
		e_w = dWeights[i];
		d2 = dDistance2(p_x, p_y, x, y);

		//if(d2 < 9.0 * h2){
			den += dGaussianKernel(h2, d2) * p_w *e_w;
		//}

		//g = dGaussianKernel(h2, d2) * p_w *e_w;
		//den += g;
	}

	x = dPoints.xCoordinates[tid];
	y = dPoints.yCoordinates[tid];
	p_w = dPoints.weights[tid];
	e_w = dWeights[tid];
	d2 = dDistance2(p_x, p_y, x, y);
	//if(d2 < 9.0 * h2){
		den_itselft = dGaussianKernel(h2, d2) * p_w *e_w;
	//}

	if(dDen0 != NULL){
		//dDen0[tid] = den;
		dDen0[tid] = logf(den);
	}

	if(dDen1 != NULL){
		//dDen1[tid] = den - den_itselft;
		dDen1[tid] = logf(den - den_itselft);
	}
}

// Density at each point under adaptive bandwidth (variable bandwidth at each point in dHs)
__global__ void DensityAtPoints(float* dHs, const SamplePoints dPoints, float *dWeights, float* dDen0, float* dDen1){
	// serial point ID
	int tid = (blockIdx.y * gridDim.y + blockIdx.x) * blockDim.x + threadIdx.x;

	int n = dPoints.numberOfPoints;
	// directly return if ID goes out of range
	if(tid >= n){
		return;
	}
	//printf("dHS[%d]: %.2f\n", tid, dHs[tid]);

	// otherwise calculate density at point ID = tid
	float p_x = dPoints.xCoordinates[tid];
	float p_y = dPoints.yCoordinates[tid];
	float den = 0.0f;

	//printf("%d %.3f %.3f\n", tid, p_x, p_y);
	int i;
	float x, y, h, p_w, e_w, d2;//, g;
	float den_itselft = 0.0f;
	for (i = 0; i < n; i++){
		x = dPoints.xCoordinates[i];
		y = dPoints.yCoordinates[i];
		p_w = dPoints.weights[i];
		e_w = dWeights[i];
		d2 = dDistance2(p_x, p_y, x, y);
		h = dHs[i];
		//if(d2 < 9.0 * h * h){
			den += dGaussianKernel(h * h, d2) * p_w *e_w;
		//}

		//den += g;

		//printf("%.2f %.2f %.2f %.2f %.2f %.2f %.2f %.2f\n", x, y, p_w, e_w, d2, h, g, den);
	}



	x = dPoints.xCoordinates[tid];
	y = dPoints.yCoordinates[tid];
	p_w = dPoints.weights[tid];
	e_w = dWeights[tid];
	d2 = dDistance2(p_x, p_y, x, y);
	h = dHs[tid];
	//if(d2 < 9.0 * h * h){
		den_itselft = dGaussianKernel(h * h, d2) * p_w *e_w;
	//}

	//printf("%.2f %.2f %.2f %.2f %.2f %.2f %.2f %.2f\n", x, y, p_w, e_w, d2, h, g, den);

	if(dDen0 != NULL){
		//dDen0[tid] = den;
		dDen0[tid] = logf(den);
		//printf("dDen0[%d]: %.5f\n", tid, dDen0[tid]);
	}

	if(dDen1 != NULL){
		//dDen1[tid] = den - den_itselft;
		dDen1[tid] = logf(den - den_itselft);
		//printf("dDen1[%d]: %.5f\n", tid, dDen1[tid]);
	}
}

// compute spatially varying bandwidths
__global__ void CalcVaryingBandwidths(const SamplePoints dPoints, float h, float * dHs)
{

	// serial point ID
	int tid = (blockIdx.y * gridDim.y + blockIdx.x) * blockDim.x + threadIdx.x;

	// directly return if ID goes out of range
	int n = dPoints.numberOfPoints;
	if(tid >= n){
		return;
	}

	// otherwise calculate varying bandwidth for point ID = tid
	dHs[tid] = h;
}

// compute spatially varying bandwidths
__global__ void CalcVaryingBandwidths(const SamplePoints dPoints, float* dDen0, float h, float alpha, float * dHs)
{

	// serial point ID
	int tid = (blockIdx.y * gridDim.y + blockIdx.x) * blockDim.x + threadIdx.x;

	// directly return if ID goes out of range
	int n = dPoints.numberOfPoints;
	if(tid >= n){
		return;
	}

	// otherwise calculate varying bandwidth for point ID = tid
	float g = expf(dReductionSum / n);
	float den = dDen0[tid];
	//if(tid == 0){
	//	den = dDen0_0;
	//}
	//float tmph = (h * (powf(expf(den) / g, alpha))); // this outmost () is NECESSARY!
	dHs[tid] = (h * (powf(expf(den) / g, alpha)));
	//printf("dHs[%d]: %4.5f \n", tid, dHs[tid]);
	//dHs[tid] = h;
}

// **===----------------- Parallel reduction (sum) ---------------------===**
//! @param g_data           input array in global memory
//                          result is expected in index 0 of g_idata
//! @param N                input number of elements to scan from input data
//! @param iteration        current iteration in reduction
// **===------------------------------------------------------------------===**
__global__ void ReductionSum(float *g_data, unsigned int N, int iteration, int num_active_items)
{
	// use shared memory
	__shared__ float s_data[BLOCK_SIZE];

	unsigned int thread_id = threadIdx.x;
	unsigned int serial_thread_id = (blockIdx.y * gridDim.y + blockIdx.x) * blockDim.x + threadIdx.x;
	// each thread loads one element from global to shared memory
	unsigned int arrIdx =  serial_thread_id * powf(BLOCK_SIZE, iteration);

	if(arrIdx < N && serial_thread_id < num_active_items){
		s_data[thread_id] = g_data[arrIdx];
	}
	else{
		s_data[thread_id] = 0.0f;
	}

	// sync threads to ensure all data are loaded into shared memory
	__syncthreads();

	// # of elements in the array to reduce
	unsigned int n_ele = BLOCK_SIZE; // initial # of elements = 1024

	// recursively reduce the array
	while(n_ele > 1){
		unsigned int m = n_ele / 2;
		if(thread_id < m){
			s_data[thread_id] += s_data[thread_id + m];
		}
		__syncthreads();
		n_ele /= 2;
	}

	// write result back to global memory
	if(thread_id == 0){
		unsigned int idx = (blockIdx.y * gridDim.y + blockIdx.x) * blockDim.x * powf(blockDim.x, iteration);
		if(idx < N){
			/*if(idx == 0){
				dDen0_0 = g_data[idx];
			}*/
			g_data[idx] = s_data[0];
		}


		if(num_active_items <= BLOCK_SIZE){
			dReductionSum = g_data[0];
		}
	}

}

#endif // #ifndef _KDE_KERNEL_H_
